/* Derived from MLIFE exercise */

/* To build:  nvcc -o main main.cu */
/* To run with a grid of 64x128: ./main 64 128 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BORN 1
#define DIES 0

#define id(r,c) ((r)*Ncolumns+(c))

/* build board */
void init(int Nrows, int Ncolumns, int **board, int **newboard, int **c_board, int **c_newboard){

  int r,c,n;

  *board    = (int*) calloc(Nrows*Ncolumns, sizeof(int));
  *newboard = (int*) calloc(Nrows*Ncolumns, sizeof(int));

  /* death at the border */
  for(r=0;r<Nrows;++r){
    (*board)[id(r,0)] = DIES;
    (*board)[id(r,Ncolumns-1)] = DIES;

    (*newboard)[id(r,0)] = DIES;
    (*newboard)[id(r,Ncolumns-1)] = DIES;
  }
  for(c=0;c<Ncolumns;++c){
    (*board)[id(0,c)] = DIES;
    (*board)[id(Nrows-1,c)] = DIES;

    (*newboard)[id(0,c)] = DIES;
    (*newboard)[id(Nrows-1,c)] = DIES;
  }

  /* random life */
  srand48(12345);
  for(r=1;r<Nrows-1;++r){
    for(c=1;c<Ncolumns-1;++c){
      double rn = drand48();
      (*board)[id(r,c)] = BORN*(rn<0.5) + DIES*(rn>=0.5);
    }
  }

  /* EX01: allocate 1D DEVICE arrays with Nrows*Ncolumns ints for c_board and c_newboard here using cudaMalloc */

  /* EX02a: copy board state from HOST board to DEVICE c_board using cudaMemcpy */

  /* EX02b: copy newboard state from HOST newboard to DEVICE c_newboard using cudaMemcpy */

}

void destroy(int *board, int *newboard){
  free(board);
  free(newboard);
}

/* EX03: convert this to a CUDA kernel */
/* EX03a: annotate to indicate a kernel */
void update(int Nrows, int Ncolumns, int *board, int *newboard){

  /* EX03b: replace double loop with 2D thread array */
  for(int r=1;r<Nrows-1;++r)
    for(int c=1;c<Ncolumns-1;++c){
      /* EX03c: convert thread indices and block indices into r,c */
      
      /* EX03d: need to make sure indices r,c are in range 1<=r<Nrows-1, 1<=c<Ncolumns-1 */
      int s = 
	board[id(r-1,c-1)]+board[id(r-1,c-0)]+board[id(r-1,c+1)]+
	board[id(r+0,c-1)]+                   board[id(r+0,c+1)]+
	board[id(r+1,c-1)]+board[id(r+1,c-0)]+board[id(r+1,c+1)];

      newboard[id(r,c)]
	= (s<2)*DIES + (s==2)*board[id(r,c)] + (s==3)*BORN + (s>3)*DIES;
    }
}

/* EX04: add a copy from DEVICE to HOST using cudaMemcpy */
void print(int Nrows, int Ncolumns, int *board, int *c_board){

  /* EX04: put cudaMemcpy here to copy from DEVICE c_board to HOST board*/


  /* No need tochange this bit */
  system("clear");
  for(int r=0;r<Nrows;++r){
    for(int c=0;c<Ncolumns;++c){
      if(board[id(r,c)]==BORN) printf("*");
      else printf(" ");
    }
    printf("\n");
  }
}


int main(int argc, char **argv){

  if(argc<3){
    printf("usage: main [Nrows] [Ncolumns]\n");
    exit(1);
  }

  /* initialize board */
  int Nrows    = atoi(argv[1]);
  int Ncolumns = atoi(argv[2]);
  int *board, *newboard;
  int *c_board, *c_newboard;
  
  init(Nrows, Ncolumns, &board, &newboard, &c_board, &c_newboard);

  /* run some iterations */
  int Nit = 100;
  for(int it=0;it<Nit;++it){
    
    /* EX05a: define thread-block size and grid size here using 16x16 thread-blocks*/
    int T = 16;
    dim3 bDim;
    dim3 gDim;

    /* EX05b: add kernel launch syntax here */
    update(Nrows, Ncolumns, c_board, c_newboard);

    /* EX05c: add kernel launch syntax here */
    update(Nrows, Ncolumns, c_newboard, c_board);
    
    print(Nrows, Ncolumns, board, c_board);
  }

  destroy(board, newboard);

  exit(0);
  return 0;
}
