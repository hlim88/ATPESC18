// to compile on cooley: nvcc  -arch sm_30 -o mandelbrot mandelbrot.cu -lm 
// to run on cooley:    ./mandelbrot 


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define MXITER 1000
#define NPOINTS 2048

// we will use these later to specify a 16x16 thread-block size
#define TX 16
#define TY 16

typedef struct {
  
  double r;
  double i;
  
}d_complex;

// return 1 if c is outside the mandelbrot set
// return 0 if c is inside the mandelbrot set

// TASK 1: annotate this as a device function 
__device__ int testpoint(d_complex c){
  
  d_complex z = c;
  
  for(int iter=0; iter<MXITER; iter++){
    
    double temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i)>4.0){
      return 1;
    }
  }
  
  return 0;
  
}

// FREEBIE: partial reduction 
__device__ void partialReduction(int outside, int *outsideCounts){
  
  __shared__ int s_outside[TX*TY];
  int t = threadIdx.x + threadIdx.y*TX;
  s_outside[t] = outside;
  
  int alive = TX*TY;
  while(alive>1){
    
    __syncthreads();
    
    alive /= 2;
    if(t<alive && t+alive<TX*TY)
      s_outside[t] += s_outside[t+alive];
    
  }
  
  if(t==0){
    int b = blockIdx.x + gridDim.x*blockIdx.y;
    outsideCounts[b] = s_outside[0];
  }
}

// TASK 2: make this a kernel that processes 
// (i,j) \in   [blockIdx.x*blockDim.x,(blockIdx.x+1)*blockDim.x) 
//           x [blockIdx.y*blockDim.y,(blockIdx.y+1)*blockDim.y) 

// TASK 2a: annotate this to indicate it is a kernel and change return type to void
__global__ void mandeloutside(int * outsideCounts){

  double eps = 1e-5;

  d_complex c;

  // TASK 2b: replace loop structures with (i,j) defined from blockIdx, blockDim, threadIdx
  //  for(i=0;i<NPOINTS;i++){
  //    for(j=0;j<NPOINTS;j++){
  int i = threadIdx.x+blockIdx.x*TX; 
  int j = threadIdx.y+blockIdx.y*TY; 

  c.r = -2. + 2.5*((double)i)/(double)(NPOINTS)+eps;
  c.i =       1.125*((double)j)/(double)(NPOINTS)+eps;
  
  // TASK 2c: replace this with a partial sum reduction of numoutside in thread block
  int outside = 0; 
  if(i<NPOINTS && j<NPOINTS){
    outside = testpoint(c);
  }
  //   }
  // }

  // FREEBIE: reduction of TX*TY values to one value on each thread-block
  partialReduction(outside, outsideCounts);

}

int main(int argc, char **argv){

  // TASK 3a: define dim3 variables for the grid size and thread-block size
  int GX = (NPOINTS+TX-1)/TX;
  int GY = (NPOINTS+TY-1)/TY;
  dim3 dimGrid(GX,GY,1);
  dim3 dimBlock(TX,TY,1);

  // TASK 3b: use cudaMalloc to create a DEVICE array that has one entry for each thread-block
  int *c_outsideCounts;
  hipMalloc(&c_outsideCounts, GX*GY*sizeof(int));

  // FREEBIE: create CUDA events for timing
  hipEvent_t start, end;

  hipEventCreate(&start);
  hipEventCreate(&end);
  
  hipEventRecord(start);
  
  // TASK 3c: replace this with a kernel call
  mandeloutside <<< dimGrid, dimBlock >>> (c_outsideCounts);
  
  // FREEBIE: timing
  float elapsed;
  hipEventRecord(end);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed, start, end);
  elapsed /= 1000;
  printf("elapsed = %g\n", elapsed);

  // TASK 3d: allocate a HOST array to receive the contents of the c_outsideCounts array
  int *h_outsideCounts = (int*) calloc(GX*GY, sizeof(int));
  
  // TASK 3e: use cudaMemcpy to copy the contents of the entries of c_outsideCounts to h_outsideCounts
  hipMemcpy(h_outsideCounts, c_outsideCounts, GX*GY*sizeof(int), hipMemcpyDeviceToHost);

  // TASK 3f: sum up the outsideCounts 
  int numoutside = 0;
  for(int n=0;n<GX*GY;++n){
    numoutside += h_outsideCounts[n];
  }

  printf("numoustide = %d\n", numoutside);

  double area = 2.*2.5*1.125*(NPOINTS*NPOINTS-numoutside)/(NPOINTS*NPOINTS);

  printf("area = %17.15lf\n", area);

  return 0;
}  
