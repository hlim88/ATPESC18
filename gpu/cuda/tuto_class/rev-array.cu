#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void revArray(int N, float *a, float *b) {

	int n = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(n<N) {
	 b[N-1-n] = a[n];
	}

}


int main(int argc, char **argv) {

	int N = 100;

	//Host memory allocation
	float *h_a = (float*) malloc(N*sizeof(float));
	float *h_b = (float*) malloc(N*sizeof(float));
	int n;

	for(n=0;n<N;n++) {
 	  h_a[n] = 1+n;
	}

	// Device memory allocation
	float *d_a, *d_b;

	hipMalloc(&d_a, N*sizeof(float));
	hipMalloc(&d_b, N*sizeof(float));

	// Copy data from host to device
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	
	//save this for later
	int NthreadsPerBlock = 10;
	int NthreadBlocks = (N+NthreadsPerBlock-1)/NthreadsPerBlock ;
	revArray<<<NthreadBlocks, NthreadsPerBlock>>>(N,d_a,d_b);

	//copy result from device to host
	hipMemcpy(h_b, d_b, N*sizeof(float), hipMemcpyDeviceToHost);


	for(n=0;n<N;++n) {
	 printf("h_b[%d] = %g\n",n,h_b[n]);
	}

	free(h_a);
	free(h_b);

	hipFree(d_a);
	hipFree(d_b);

	return 0;

}
