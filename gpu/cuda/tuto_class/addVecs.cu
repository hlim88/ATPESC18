#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void addVectors(int N, float *a, float *b, float *c) {

	int n = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(n<N) {
	 c[n] = a[n] + b[n];
	}

}


int main(int argc, char **argv) {

	int N = 100;

	//Host memory allocation
	float *h_a = (float*) malloc(N*sizeof(float));
	float *h_b = (float*) malloc(N*sizeof(float));
	float *h_c = (float*) malloc(N*sizeof(float));

	int n;

	for(n=0;n<N;n++) {
 	  h_a[n] = 1+n;
	  h_b[n] = 1-n;
	}

	// Device memory allocation
	float *d_a, *d_b, *d_c;

	hipMalloc(&d_a, N*sizeof(float));
	hipMalloc(&d_b, N*sizeof(float));
	hipMalloc(&d_c, N*sizeof(float));

	// Copy data from host to device
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);
	
	//save this for later
	int NthreadsPerBlock = 10;
	int NthreadBlocks = (N+NthreadsPerBlock-1)/NthreadsPerBlock ;
	addVectors<<<NthreadBlocks, NthreadsPerBlock>>>(N,d_a,d_b,d_c);

	//copy result from device to host
	hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);


	for(n=0;n<5;++n) {
	 printf("h_c[%d] = %g\n",n,h_c[n]);
	}

	free(h_a);
	free(h_b);
	free(h_b);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}
