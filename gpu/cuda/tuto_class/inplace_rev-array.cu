#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

__global__ void revArray(int N, float *a) {

	int n = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(n<N/2) {
	  float a1 = a[n];
	  a[n] = a[N-1-n];
	  a[N-1-n] = a1;
	}

}


int main(int argc, char **argv) {

	int N = 100;

	//Host memory allocation
	float *h_a = (float*) malloc(N*sizeof(float));
	float *h_b = (float*) malloc(N*sizeof(float));
	int n;

	for(n=0;n<N;n++) {
 	  h_a[n] = 1+n;
	}

	// Device memory allocation
	float *d_a;

	hipMalloc(&d_a, N*sizeof(float));

	// Copy data from host to device
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	
	//save this for later
	int NthreadsPerBlock = 10;
	int NthreadBlocks = ((N/2)+NthreadsPerBlock-1)/NthreadsPerBlock ;
	revArray<<<NthreadBlocks, NthreadsPerBlock>>>(N,d_a);

	//copy result from device to host
	hipMemcpy(h_a, d_a, N*sizeof(float), hipMemcpyDeviceToHost);


	for(n=0;n<N;++n) {
	 printf("h_a[%d] = %g\n",n,h_a[n]);
	}

	free(h_a);

	hipFree(d_a);

	return 0;

}
